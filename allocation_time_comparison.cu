#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>

static inline void
checkDrvError(hipError_t res, const char *tok, const char *file, unsigned line)
{
    if (res != hipSuccess) {
        const char *errStr = NULL;
        (void)hipDrvGetErrorString(res, &errStr);
        std::cerr << file << ':' << line << ' ' << tok
                  << "failed (" << (unsigned)res << "): " << errStr << std::endl;
        abort();
    }
}

#define CHECK_DRV(x) checkDrvError(x, #x, __FILE__, __LINE__);

typedef struct _vmm_struct {
	hipMemGenericAllocationHandle_t handle;
	hipDeviceptr_t ptr;
	size_t padded_size;
} vmm_struct;

vmm_struct low_level_allocation (size_t size) {
	vmm_struct vmm;
	size_t granularity;
	hipMemAllocationProp prop = {};
    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
	int cur_device = 0;
	hipGetDevice(&cur_device);
    prop.location.id = cur_device;
    CHECK_DRV(hipMemGetAllocationGranularity(&granularity, &prop, hipMemAllocationGranularityMinimum));
    // printf("granularity: %ld\n", granularity);
	vmm.padded_size = ((size+ granularity - 1) / granularity) * granularity;
    // printf("padded size: %ld\n", padded_size);
	CHECK_DRV(hipMemCreate(&vmm.handle, vmm.padded_size, &prop, 0));
	CHECK_DRV(hipMemAddressReserve(&vmm.ptr, vmm.padded_size, 0, 0, 0));
	CHECK_DRV(hipMemMap(vmm.ptr, vmm.padded_size, 0, vmm.handle, 0));
	hipMemAccessDesc accessDesc = {};
    accessDesc.location.type = hipMemLocationTypeDevice;
    accessDesc.location.id = 0;
    accessDesc.flags = hipMemAccessFlagsProtReadWrite;
    CHECK_DRV( hipMemSetAccess(vmm.ptr, vmm.padded_size , &accessDesc, 1ULL) );

	return vmm;
}

void low_level_free (vmm_struct vmm) {
	CHECK_DRV(hipMemUnmap(vmm.ptr, vmm.padded_size));
	CHECK_DRV(hipMemRelease(vmm.handle));
	CHECK_DRV(hipMemAddressFree(vmm.ptr, vmm.padded_size));
}

int main(int argc, char* argv[]) {
    size_t size = 
}