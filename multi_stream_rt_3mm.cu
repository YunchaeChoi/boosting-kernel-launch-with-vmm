#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <bemps.hpp>
#include <iostream>
// #include <thread>
#include <pthread.h>
#include <unistd.h>
#include "../../common/polybenchUtilFuncts.h"

#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

/* Problem size. */
/* default setting */
// # define NI 4096
// # define NJ 4096
// # define NK 4096
// # define NL 4096
// # define NM 4096

# define NI 4096
# define NJ 4096
# define NK 4096
# define NL 4096
# define NM 4096

#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

typedef float DATA_TYPE;

static inline void
checkRtError(hipError_t res, const char *tok, const char *file, unsigned line)
{
    if (res != hipSuccess) {
        std::cerr << file << ':' << line << ' ' << tok
                  << "failed (" << (unsigned)res << "): " << hipGetErrorString(res) << std::endl;
        abort();
    }
}

#define CHECK_RT(x) checkRtError(x, #x, __FILE__, __LINE__);

static inline void
checkDrvError(hipError_t res, const char *tok, const char *file, unsigned line)
{
    if (res != hipSuccess) {
        const char *errStr = NULL;
        (void)hipDrvGetErrorString(res, &errStr);
        std::cerr << file << ':' << line << ' ' << tok
                  << "failed (" << (unsigned)res << "): " << errStr << std::endl;
        abort();
    }
}

#define CHECK_DRV(x) checkDrvError(x, #x, __FILE__, __LINE__);

typedef struct _vmm_struct {
	hipMemGenericAllocationHandle_t handle;
	hipDeviceptr_t ptr;
	size_t padded_size;
} vmm_struct;

vmm_struct low_level_allocation (size_t size) {
	vmm_struct vmm;
	size_t granularity;
	hipMemAllocationProp prop = {};
    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
	int cur_device = 0;
	hipGetDevice(&cur_device);
    prop.location.id = cur_device;
    CHECK_DRV(hipMemGetAllocationGranularity(&granularity, &prop, hipMemAllocationGranularityMinimum));
    // printf("granularity: %ld\n", granularity);
	vmm.padded_size = ((size+ granularity - 1) / granularity) * granularity;
    // printf("padded size: %ld\n", padded_size);
	CHECK_DRV(hipMemCreate(&vmm.handle, vmm.padded_size, &prop, 0));
	CHECK_DRV(hipMemAddressReserve(&vmm.ptr, vmm.padded_size, 0, 0, 0));
	CHECK_DRV(hipMemMap(vmm.ptr, vmm.padded_size, 0, vmm.handle, 0));
	hipMemAccessDesc accessDesc = {};
    accessDesc.location.type = hipMemLocationTypeDevice;
    accessDesc.location.id = 0;
    accessDesc.flags = hipMemAccessFlagsProtReadWrite;
    CHECK_DRV( hipMemSetAccess(vmm.ptr, vmm.padded_size , &accessDesc, 1ULL) );

	return vmm;
}

void low_level_free (vmm_struct vmm) {
	CHECK_DRV(hipMemUnmap(vmm.ptr, vmm.padded_size));
	CHECK_DRV(hipMemRelease(vmm.handle));
	CHECK_DRV(hipMemAddressFree(vmm.ptr, vmm.padded_size));
}

void init_array(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D)
{
	int i, j;

	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NK; j++)
		{
			A[i*NK + j] = ((DATA_TYPE) i*j) / NI;
			// printf("A[%d]: %f\n", i*NK + j, A[i*NK + j]);
		}
	}
  
	for (i = 0; i < NK; i++)
	{
		for (j = 0; j < NJ; j++)
		{
			B[i*NJ + j] = ((DATA_TYPE) i*(j+1)) / NJ;
		}
	}
  
	for (i = 0; i < NJ; i++)
	{
		for (j = 0; j < NM; j++)
		{
			C[i*NM + j] = ((DATA_TYPE) i*(j+3)) / NL;
		}
	}
  
	for (i = 0; i < NM; i++)
	{
		for (j = 0; j < NL; j++)
		{
			D[i*NL + j] = ((DATA_TYPE) i*(j+2)) / NK;
		}
	}
}


void compareResults(DATA_TYPE *G, DATA_TYPE *G_outputFromGpu)
{
	int i,j,fail;
	fail = 0;

	for (i=0; i < NI; i++)
	{
		for (j=0; j < NL; j++)
		{
			if (percentDiff(G[i*NL + j], G_outputFromGpu[i*NL + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
			{
				fail++;				
			}
		}
	}
	
	// print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


	
__global__ void mm3_kernel1(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *E)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NI) && (j < NJ))
	{
		int k;
		for(k=0; k < NK; k++)
		{
			E[i * NJ + j] += A[i * NK + k] * B[k * NJ + j];
			// printf("%f\n", B[k * NJ + j]);
			// printf("E[%d]: %f\n", i * NJ + j, E[i * NJ + j]);
		}
	}
}

	
__global__ void mm3_kernel2(DATA_TYPE *C, DATA_TYPE *D, DATA_TYPE *F)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NJ) && (j < NL))
	{
		int k;
		for(k=0; k < NM; k++)
		{
			F[i * NL + j] += C[i * NM + k] * D[k * NL +j];
		}
	}
}

	
__global__ void mm3_kernel3(DATA_TYPE *E, DATA_TYPE *F, DATA_TYPE *G)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NI) && (j < NL))
	{
		int k;
		for(k=0; k < NJ; k++)
		{
			G[i * NL + j] += E[i * NJ + k] * F[k * NL + j];
			// printf("%f\n", G[i * NL + j]);
		}
	}
}


void mm3_cpu(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C, DATA_TYPE *D, DATA_TYPE *E, DATA_TYPE *F, DATA_TYPE *G)
{
	int i,j,k;
	
	/* E := A*B */
	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NJ; j++)
		{
			E[i*NJ + j] = 0;
			for (k = 0; k < NK; ++k)
			{
				E[i*NJ + j] += A[i*NK + k] * B[k*NJ + j];
			}
		}
	}
		
	/* F := C*D */
	for (i = 0; i < NJ; i++)
	{
		for (j = 0; j < NL; j++)
		{
			F[i*NL + j] = 0;
			for (k = 0; k < NM; ++k)
			{
				F[i*NL + j] += C[i*NM + k] * D[k*NL + j];
			}
		}
	}

  	/* G := E*F */
	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NL; j++)
		{
			G[i*NL + j] = 0;
			for (k = 0; k < NJ; ++k)
			{
				G[i*NL + j] += E[i*NJ + k] * F[k*NL + j];
			}
		}
	}
}

void *free_in_thread(void *A) {
	hipFree((DATA_TYPE*)A);
	return NULL;
}


int main(int argc, char* argv[]) {

	

	double t_start, t_end;
	t_start = rtclock();
	CHECK_RT(hipFree(0));

	pthread_t tid[4];

	hipStream_t stream[3];
	for (int i=0; i<3; i++) {
		hipStreamCreate(&stream[i]);
	}

	
	size_t size = NI * NK * sizeof(DATA_TYPE);

	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid1((size_t)(ceil( ((float)NJ) / ((float)DIM_THREAD_BLOCK_X) )),(size_t)(ceil((float)NI/ ((float)DIM_THREAD_BLOCK_Y) )));
	dim3 grid2((size_t)(ceil( ((float)NL) / ((float)DIM_THREAD_BLOCK_X) )),(size_t)(ceil((float)NJ/ ((float)DIM_THREAD_BLOCK_Y) )));
	dim3 grid3((size_t)(ceil( ((float)NL) / ((float)DIM_THREAD_BLOCK_X) )),(size_t)(ceil((float)NI/ ((float)DIM_THREAD_BLOCK_Y) )));

	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);

	DATA_TYPE* A;
	DATA_TYPE* B;
	DATA_TYPE* C;
	DATA_TYPE* D;
	DATA_TYPE* E;
	DATA_TYPE* F;
	DATA_TYPE* G;
	DATA_TYPE* G_outputFromGpu;

	A = (DATA_TYPE*)malloc(NI*NK*sizeof(DATA_TYPE));
	B = (DATA_TYPE*)malloc(NK*NJ*sizeof(DATA_TYPE));
	C = (DATA_TYPE*)malloc(NJ*NM*sizeof(DATA_TYPE));
	D = (DATA_TYPE*)malloc(NM*NL*sizeof(DATA_TYPE));
	E = (DATA_TYPE*)malloc(NI*NJ*sizeof(DATA_TYPE));
	F = (DATA_TYPE*)malloc(NJ*NL*sizeof(DATA_TYPE));
	G = (DATA_TYPE*)malloc(NI*NL*sizeof(DATA_TYPE));
	G_outputFromGpu = (DATA_TYPE*)malloc(NI*NL*sizeof(DATA_TYPE));

	init_array(A, B, C, D);

	

	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *C_gpu;
	DATA_TYPE *D_gpu;
	DATA_TYPE *E_gpu;
	DATA_TYPE *F_gpu;
	DATA_TYPE *G_gpu;

	hipMalloc((void **)&A_gpu, size);
	hipMalloc((void **)&B_gpu, size);
	hipMalloc((void **)&C_gpu, size);
	hipMalloc((void **)&D_gpu, size);
	hipMalloc((void **)&E_gpu, size);
	hipMalloc((void **)&F_gpu, size);
	hipMalloc((void **)&G_gpu, size);
	
	hipMemcpyAsync(A_gpu, A, sizeof(DATA_TYPE) * NI * NK, hipMemcpyHostToDevice, stream[0]);
	hipMemcpyAsync(B_gpu, B, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice, stream[0]);

	mm3_kernel1<<<grid1,block, 0, stream[0]>>>(A_gpu, B_gpu, E_gpu);
	
	hipMemcpyAsync(C_gpu, C, sizeof(DATA_TYPE) * NJ * NM, hipMemcpyHostToDevice, stream[1]);
	hipMemcpyAsync(D_gpu, D, sizeof(DATA_TYPE) * NM * NL, hipMemcpyHostToDevice, stream[1]);

	hipEventRecord(start);

	hipStreamSynchronize(stream[0]);
	if (pthread_create(&tid[0], NULL, free_in_thread, (void*)A_gpu) < 0) {
		perror("pthread 0 create error\n");
		exit(EXIT_FAILURE);
	}
	if (pthread_create(&tid[1], NULL, free_in_thread, (void*)B_gpu) < 0) {
		perror("pthread 0 create error\n");
		exit(EXIT_FAILURE);
	}
	// hipFree(A_gpu);
	// hipFree(B_gpu);

	// mm3_kernel1<<<grid1,block>>>(A_gpu, B_gpu, E_gpu);
	// mm3_kernel1<<<grid1,block, 0, stream[0]>>>(A_gpu, B_gpu, E_gpu);
	// hipDeviceSynchronize();
	mm3_kernel2<<<grid2,block, 0, stream[1]>>>(C_gpu, D_gpu, F_gpu);
	
	// pthread_create(&tid[0], NULL, free_in_thread, &VMM_A);
	// pthread_create(&tid[1], NULL, free_in_thread, &VMM_B);
	
	
	// mm3_kernel2<<<grid2,block>>>(C_gpu, D_gpu, F_gpu);
	// mm3_kernel2<<<grid2,block, 0, stream[1]>>>(C_gpu, D_gpu, F_gpu);
	// hipDeviceSynchronize();
	hipStreamSynchronize(stream[1]);
	hipDeviceSynchronize();

	// pthread_create(&tid[2], NULL, free_in_thread, &VMM_C);
	// pthread_create(&tid[3], NULL, free_in_thread, &VMM_D);
	if (pthread_create(&tid[0], NULL, free_in_thread, (void*)B_gpu) < 0) {
		perror("pthread 0 create error\n");
		exit(EXIT_FAILURE);
	}
	if (pthread_create(&tid[1], NULL, free_in_thread, (void*)D_gpu) < 0) {
		perror("pthread 0 create error\n");
		exit(EXIT_FAILURE);
	}
	// hipFree(C_gpu);
	// hipFree(D_gpu);
	// mm3_kernel3<<<grid3,block>>>(E_gpu, F_gpu, G_gpu);
	mm3_kernel3<<<grid3,block, 0, stream[2]>>>(E_gpu, F_gpu, G_gpu);
	// hipDeviceSynchronize();
	hipStreamSynchronize(stream[2]);

	hipFree(E_gpu);
	hipFree(F_gpu);
	hipFree(G_gpu);
	
	hipEventRecord(end);
	hipEventSynchronize(end);

	float milliseconds = 0.0;
	hipEventElapsedTime(&milliseconds, start, end);
	// printf("Elapsed Time (by cudaEvent): %fms\n", milliseconds);
	printf("%f", milliseconds);
	
	// hipMemcpy(G_outputFromGpu, G_gpu, size, hipMemcpyDeviceToHost);

	// mm3_cpu(A, B, C, D, E, F, G);
	// compareResults(G, G_outputFromGpu);

	for (int i=0; i < size/sizeof(DATA_TYPE); i++) {
		// printf("G, G_outputFromGpu (%d): %f, %f\n", i, G[i], G_outputFromGpu[i]);
	}

	// low_level_free(VMM_A);
	// low_level_free(VMM_B);
	// low_level_free(VMM_C);
	// low_level_free(VMM_D);
	/*
		low_level_free(VMM_E);
		low_level_free(VMM_F);
		low_level_free(VMM_G);
	*/

	// hipEventRecord(end);
	// hipEventSynchronize(end);

	// float milliseconds = 0.0;
	// hipEventElapsedTime(&milliseconds, start, end);
	// printf("Elapsed Time (by cudaEvent): %fms\n", milliseconds);

	free(A);
	free(B);
	free(C);
	free(D);
	free(E);
	free(F);
	free(G);
	free(G_outputFromGpu);

	hipEventDestroy(start);
	hipEventDestroy(end);
	
	t_end = rtclock();

	// for (int i=0;i <3;i++) {
	// 	hipStreamDestroy(streams[i]);
	// }

	for(int i=0;i<3;i++) {
		hipStreamDestroy(stream[i]);
	}
	

	// printf("Total Elapsed time (by gettimeofday): %.6lfs\n", t_end - t_start);

	return 0;
}