#include <stdio.h>
#include <stdlib.h>
#include <low.h>

int main() {
    hipFree(0);

    size_t size = 16384 * 16384 * 4; // 1 GiB
    float* d_A;
    hipMalloc(&d_A, size);
    hipError_t err = hipGetLastError();
    printf("%s\n", hipGetErrorString(err));
    vmm_struct d_B =  low_level_allocation(size);

    hipEvent_t start;
    hipEvent_t end;

    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
    low_level_free(d_B);
    hipEventRecord(end);
    hipEventSynchronize(end);

    float milliseconds = 0.0;
    hipEventElapsedTime(&milliseconds, start, end);

    printf("VMM: %fms\n", milliseconds);

    hipEventDestroy(start);
    hipEventDestroy(end);

    hipFree(d_A);
    return 0;
}

